#include "hip/hip_runtime.h"
#include <hip/hip_fp8.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n", file, line,
                static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}

// 使用CUDA 12.8的FP8 Tensor Core API
__global__ void fp8_tensorcore_matmul(const __hip_fp8_e4m3_fnuz* A, const __hip_fp8_e4m3_fnuz* B, __hip_fp8_e4m3_fnuz* C) {
    // 每个线程块处理整个8x8矩阵
    int row = threadIdx.y;
    int col = threadIdx.x;
    
    float sum = 0.0f;
    
    for (int k = 0; k < 8; ++k) {
        // 使用CUDA 12.8的FP8转换函数
        float a = __half2float(__hip_cvt_fp8_to_halfraw(*reinterpret_cast<const __hip_fp8_storage_t*>(&A[row * 8 + k]), __HIP_E4M3_FNUZ));
        float b = __half2float(__hip_cvt_fp8_to_halfraw(*reinterpret_cast<const __hip_fp8_storage_t*>(&B[k * 8 + col]), __HIP_E4M3_FNUZ));
        sum += a * b;
    }
    
    // 将结果转换回FP8
    __hip_fp8_storage_t fp8_result = __hip_cvt_halfraw_to_fp8(__float2half(sum), __HIP_E4M3_FNUZ, __HIP_SATFINITE);
    C[row * 8 + col] = *reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&fp8_result);
}

int main() {
    const int M = 8, N = 8, K = 8;
    
    // 初始化主机端数据
    __hip_fp8_e4m3_fnuz A_host[M*K], B_host[K*N], C_host[M*N];
    
    // 填充测试数据
    for (int i = 0; i < M*K; ++i) {
        float val = (i % 3) * 0.5f;
        __hip_fp8_storage_t fp8_val = __hip_cvt_halfraw_to_fp8(__float2half(val), __HIP_E4M3_FNUZ, __HIP_SATFINITE);
        A_host[i] = *reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&fp8_val);
    }
    
    for (int i = 0; i < K*N; ++i) {
        float val = (i % 5) * 0.3f;
        __hip_fp8_storage_t fp8_val = __hip_cvt_halfraw_to_fp8(__float2half(val), __HIP_E4M3_FNUZ, __HIP_SATFINITE);
        B_host[i] = *reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&fp8_val);
    }
    
    // 分配设备内存
    __hip_fp8_e4m3_fnuz *A_dev, *B_dev, *C_dev;
    CHECK_CUDA_ERROR(hipMalloc(&A_dev, M*K*sizeof(__hip_fp8_e4m3_fnuz)));
    CHECK_CUDA_ERROR(hipMalloc(&B_dev, K*N*sizeof(__hip_fp8_e4m3_fnuz)));
    CHECK_CUDA_ERROR(hipMalloc(&C_dev, M*N*sizeof(__hip_fp8_e4m3_fnuz)));
    
    // 拷贝数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(A_dev, A_host, M*K*sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(B_dev, B_host, K*N*sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyHostToDevice));
    
    // 启动核函数 - 使用8x8线程块
    dim3 block(8, 8);
    fp8_tensorcore_matmul<<<1, block>>>(A_dev, B_dev, C_dev);
    
    // 拷贝结果回主机
    CHECK_CUDA_ERROR(hipMemcpy(C_host, C_dev, M*N*sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyDeviceToHost));
    
    // 打印结果
    std::cout << "FP8 8x8 Matrix Multiplication Result:\n";
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float val = __half2float(__hip_cvt_fp8_to_halfraw(*reinterpret_cast<const __hip_fp8_storage_t*>(&C_host[i*N + j]), __HIP_E4M3_FNUZ));
            printf("%.2f ", val);
        }
        printf("\n");
    }
    
    // 释放设备内存
    CHECK_CUDA_ERROR(hipFree(A_dev));
    CHECK_CUDA_ERROR(hipFree(B_dev));
    CHECK_CUDA_ERROR(hipFree(C_dev));
    
    return 0;
}