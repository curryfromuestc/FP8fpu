#include <iostream>                                           // 标准输入输出流
#include <cutlass/cutlass.h>                       // 引入cutlass头文件
#include <cutlass/numeric_types.h>                 // For FP8 types
#include <cutlass/gemm/device/gemm.h>              // For GEMM operation
#include <cutlass/gemm/device/gemm_universal.h>    // For universal GEMM
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <hip/hip_runtime.h>                          // For CUDA runtime API

using ElementA = cutlass::float_e4m3_t;
using ElementB = cutlass::float_e4m3_t;
using ElementOutput = cutlass::float_e4m3_t;
using ElementAccumulator = float;

// Define the GEMM operation using Tensor Core
using Gemm = cutlass::gemm::device::GemmUniversalAdapter<
    cutlass::gemm::device::GemmUniversal<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementOutput, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,  // Use Tensor Core
        cutlass::arch::Sm89,            // Ampere architecture
        cutlass::gemm::GemmShape<128, 128, 32>,  // Tile size optimized for FP8
        cutlass::gemm::GemmShape<64, 64, 32>,    // Threadblock tile size
        cutlass::gemm::GemmShape<16, 8, 32>,     // Warp tile size
        cutlass::epilogue::thread::LinearCombination<
            ElementOutput,
            1,
            ElementAccumulator,
            ElementAccumulator
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2,
        cutlass::arch::OpMultiplyAdd
    >
>;

void generate_tensor_1D(ElementA *A, int M) {
    for (int i = 0; i < M; i++) {
        // Convert random integer to FP8 by creating a proper FP8 value
        float value = static_cast<float>(rand() % 100) / 100.0f;
        A[i] = cutlass::float_e4m3_t(value);
    }
}

int main() {
    int M = 128;
    float scale = 0.25*0.25;

    // Allocate device memory for matrices
    ElementA *d_A = nullptr;
    ElementB *d_B = nullptr;
    ElementOutput *d_C = nullptr;
    
    hipError_t cuda_status;
    
    // Allocate device memory
    cuda_status = hipMalloc(&d_A, M * M * sizeof(ElementA));
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to allocate device memory for A: " << hipGetErrorString(cuda_status) << std::endl;
        return -1;
    }
    
    cuda_status = hipMalloc(&d_B, M * M * sizeof(ElementB));
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to allocate device memory for B: " << hipGetErrorString(cuda_status) << std::endl;
        hipFree(d_A);
        return -1;
    }
    
    cuda_status = hipMalloc(&d_C, M * M * sizeof(ElementOutput));
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to allocate device memory for C: " << hipGetErrorString(cuda_status) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        return -1;
    }

    // Allocate and initialize host memory
    ElementA *h_A = (ElementA *)malloc(M * M * sizeof(ElementA));
    ElementB *h_B = (ElementB *)malloc(M * M * sizeof(ElementB));
    ElementOutput *h_C = (ElementOutput *)malloc(M * M * sizeof(ElementOutput));
    
    if (!h_A || !h_B || !h_C) {
        std::cerr << "Failed to allocate host memory" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        return -1;
    }

    // Initialize host matrices
    generate_tensor_1D(h_A, M * M);
    generate_tensor_1D(h_B, M * M);

    // Copy data to device
    cuda_status = hipMemcpy(d_A, h_A, M * M * sizeof(ElementA), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to copy A to device: " << hipGetErrorString(cuda_status) << std::endl;
        goto cleanup;
    }

    cuda_status = hipMemcpy(d_B, h_B, M * M * sizeof(ElementB), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to copy B to device: " << hipGetErrorString(cuda_status) << std::endl;
        goto cleanup;
    }

    // Initialize the GEMM operation
    Gemm gemm_op;
    
    // Set up the GEMM arguments
    typename Gemm::Arguments args(
        {M, M, M},  // problem size
        {d_A, M},   // A matrix
        {d_B, M},   // B matrix
        {d_C, M},   // C matrix
        {d_C, M},   // D matrix
        {scale},    // alpha
        {scale}     // beta
    );

    // Initialize the GEMM operation
    cutlass::Status status = gemm_op.initialize(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to initialize GEMM operation" << std::endl;
        goto cleanup;
    }

    // Run the GEMM operation
    status = gemm_op();
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to run GEMM operation" << std::endl;
        goto cleanup;
    }

    // Copy result back to host
    cuda_status = hipMemcpy(h_C, d_C, M * M * sizeof(ElementOutput), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        std::cerr << "Failed to copy result from device: " << hipGetErrorString(cuda_status) << std::endl;
        goto cleanup;
    }

    // Print some results for verification
    std::cout << "Matrix multiplication completed successfully" << std::endl;
    std::cout << "First few elements of result matrix:" << std::endl;
    for (int i = 0; i < 5; i++) {
        std::cout << static_cast<float>(h_C[i]) << " ";
    }
    std::cout << std::endl;

cleanup:
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
    
